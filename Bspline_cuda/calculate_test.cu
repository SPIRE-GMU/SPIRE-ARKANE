
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>  // time

#define N 2000
#define GRID_SIZE 11  // grid 

// compute 3-order B spline
__device__ double calculate_bspline(double *grid, double x) {
    double N0[4] = {0};
    double N1[3] = {0};
    double N2[2] = {0};
    double N3[1] = {0};

    // compute 0-order B-spline coefficients
    for (int i = 0; i <= 3; i++) {
        N0[i] = (grid[i] <= x && x < grid[i + 1]) ? 1.0 : 0.0;
    }

    // compute 1-order B-spline coefficients
    for (int i = 0; i < 3; i++) {
        double alpha_denominator = grid[i + 1] - grid[i];
        double beta_denominator = grid[i + 2] - grid[i + 1];
        double alpha = (alpha_denominator == 0) ? 0 : (x - grid[i]) / alpha_denominator * N0[i];
        double beta = (beta_denominator == 0) ? 0 : (grid[i + 2] - x) / beta_denominator * N0[i + 1];
        N1[i] = alpha + beta;
    }

    // compute 2-order B-spline coefficients
    for (int i = 0; i < 2; i++) {
        double alpha_denominator = grid[i + 2] - grid[i];
        double beta_denominator = grid[i + 3] - grid[i + 1];
        double alpha = (alpha_denominator == 0) ? 0 : (x - grid[i]) / alpha_denominator * N1[i];
        double beta = (beta_denominator == 0) ? 0 : (grid[i + 3] - x) / beta_denominator * N1[i + 1];
        N2[i] = alpha + beta;
    }

    // compute 3-order B-spline coefficients
    for (int i = 0; i < 1; i++) {
        double alpha_denominator = grid[i + 3] - grid[i];
        double beta_denominator = grid[i + 4] - grid[i + 1];
        double alpha = (alpha_denominator == 0) ? 0 : (x - grid[i]) / alpha_denominator * N2[i];
        double beta = (beta_denominator == 0) ? 0 : (grid[i + 4] - x) / beta_denominator * N2[i + 1];
        N3[i] = alpha + beta;
    }

    return N3[0];
}

// CUDA kernel
__global__ void test_kernel(double *grid, double *x, double *result) {
    // shared memory
    __shared__ double s_grid[GRID_SIZE];

    // index
    int xid = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // load grid into shared memory
    if (tid < GRID_SIZE) {
        s_grid[tid] = grid[tid];
    }
    __syncthreads();  // wait for all threads to load grid

    if (xid < N) {
        double x_val = x[xid];
        for (int i = 0; i < 7; i++) {
            result[xid * 7 + i] = calculate_bspline(s_grid + i, x_val);
        }
    }
}

int main() {
    int threads_per_block = 256;
    int number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

    // define grid
    double h_grid[GRID_SIZE] = {0};  // grid 
    for (int i = 0; i < GRID_SIZE; i++) {
        h_grid[i] = -1.0 + i * (2.0 / (GRID_SIZE - 1));
    }

    double h_x[N] = {0};  // x values
    for (int i = 0; i < N; ++i) {
        h_x[i] = (float)rand() / RAND_MAX * 2.0f - 1.0f;
    }

    double h_result[N * 7] = {0.0};  // result

    // shared memory
    double *d_grid, *d_result, *d_x;
    hipMalloc((void**)&d_grid, sizeof(h_grid));
    hipMalloc((void**)&d_result, sizeof(h_result));
    hipMalloc((void**)&d_x, sizeof(h_x));

    // copy data to device
    hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice);
    hipMemcpy(d_grid, h_grid, sizeof(h_grid), hipMemcpyHostToDevice);

    // start_time
    auto start_time = std::chrono::high_resolution_clock::now();

    // run kernel
    for (int j = 0; j < 1000; j++) {
        test_kernel<<<number_of_blocks, threads_per_block>>>(d_grid, d_x, d_result);
        hipDeviceSynchronize();
    }

    // end_time
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> time_spent = end_time - start_time;
    printf("Execution time: %.6f seconds\n", time_spent.count());

    // copy result back to host
    hipMemcpy(h_result, d_result, sizeof(double) * N * 7, hipMemcpyDeviceToHost);

    // 打印部分结果
    // for (int i = 0; i < 1000; ++i) {
    //     printf("B-spline values for x = %.2f:\n", h_x[i]);
    //     for (int j = 0; j < 7; ++j) {
    //         printf("  B[%d] = %f\n", j, h_result[i * 7 + j]);
    //     }
    // }

    // release device memory
    hipFree(d_grid);
    hipFree(d_x);
    hipFree(d_result);

    return 0;
}
