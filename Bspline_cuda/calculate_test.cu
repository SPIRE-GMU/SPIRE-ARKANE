
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>  // 高精度计时

#define N 2000
#define GRID_SIZE 11  // grid 数组的大小

// 设备函数：计算 B 样条基函数
__device__ double calculate_bspline(double *grid, double x) {
    double N0[4] = {0};
    double N1[3] = {0};
    double N2[2] = {0};
    double N3[1] = {0};

    // 计算0阶B样条系数
    for (int i = 0; i <= 3; i++) {
        N0[i] = (grid[i] <= x && x < grid[i + 1]) ? 1.0 : 0.0;
    }

    // 计算1阶B样条系数
    for (int i = 0; i < 3; i++) {
        double alpha_denominator = grid[i + 1] - grid[i];
        double beta_denominator = grid[i + 2] - grid[i + 1];
        double alpha = (alpha_denominator == 0) ? 0 : (x - grid[i]) / alpha_denominator * N0[i];
        double beta = (beta_denominator == 0) ? 0 : (grid[i + 2] - x) / beta_denominator * N0[i + 1];
        N1[i] = alpha + beta;
    }

    // 计算2阶B样条系数
    for (int i = 0; i < 2; i++) {
        double alpha_denominator = grid[i + 2] - grid[i];
        double beta_denominator = grid[i + 3] - grid[i + 1];
        double alpha = (alpha_denominator == 0) ? 0 : (x - grid[i]) / alpha_denominator * N1[i];
        double beta = (beta_denominator == 0) ? 0 : (grid[i + 3] - x) / beta_denominator * N1[i + 1];
        N2[i] = alpha + beta;
    }

    // 计算3阶B样条系数
    for (int i = 0; i < 1; i++) {
        double alpha_denominator = grid[i + 3] - grid[i];
        double beta_denominator = grid[i + 4] - grid[i + 1];
        double alpha = (alpha_denominator == 0) ? 0 : (x - grid[i]) / alpha_denominator * N2[i];
        double beta = (beta_denominator == 0) ? 0 : (grid[i + 4] - x) / beta_denominator * N2[i + 1];
        N3[i] = alpha + beta;
    }

    return N3[0];
}

// 测试内核：调用 calculate_bspline 并将结果存储到全局内存
__global__ void test_kernel(double *grid, double *x, double *result) {
    // 定义共享内存
    __shared__ double s_grid[GRID_SIZE];

    // 线程索引
    int xid = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // 将 grid 数据加载到共享内存
    if (tid < GRID_SIZE) {
        s_grid[tid] = grid[tid];
    }
    __syncthreads();  // 确保所有线程都完成了共享内存的加载

    if (xid < N) {
        double x_val = x[xid];
        for (int i = 0; i < 7; i++) {
            result[xid * 7 + i] = calculate_bspline(s_grid + i, x_val);
        }
    }
}

int main() {
    int threads_per_block = 256;
    int number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

    // 定义 grid 数组和 x 值
    double h_grid[GRID_SIZE] = {0};  // grid 数组
    for (int i = 0; i < GRID_SIZE; i++) {
        h_grid[i] = -1.0 + i * (2.0 / (GRID_SIZE - 1));
    }

    double h_x[N] = {0};  // 要计算 B 样条基函数的点
    for (int i = 0; i < N; ++i) {
        h_x[i] = (float)rand() / RAND_MAX * 2.0f - 1.0f;
    }

    double h_result[N * 7] = {0.0};  // 存储结果的变量

    // 分配设备内存
    double *d_grid, *d_result, *d_x;
    hipMalloc((void**)&d_grid, sizeof(h_grid));
    hipMalloc((void**)&d_result, sizeof(h_result));
    hipMalloc((void**)&d_x, sizeof(h_x));

    // 将数据复制到设备
    hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice);
    hipMemcpy(d_grid, h_grid, sizeof(h_grid), hipMemcpyHostToDevice);

    // 计时开始
    auto start_time = std::chrono::high_resolution_clock::now();

    // 运行测试内核 1000 次
    for (int j = 0; j < 1000; j++) {
        test_kernel<<<number_of_blocks, threads_per_block>>>(d_grid, d_x, d_result);
        hipDeviceSynchronize();
    }

    // 计时结束
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> time_spent = end_time - start_time;
    printf("Execution time: %.6f seconds\n", time_spent.count());

    // 将结果复制回主机
    hipMemcpy(h_result, d_result, sizeof(double) * N * 7, hipMemcpyDeviceToHost);

    // 打印部分结果
    // for (int i = 0; i < 10; ++i) {
    //     printf("B-spline values for x = %.2f:\n", h_x[i]);
    //     for (int j = 0; j < 7; ++j) {
    //         printf("  B[%d] = %f\n", j, h_result[i * 7 + j]);
    //     }
    // }

    // 释放设备内存
    hipFree(d_grid);
    hipFree(d_x);
    hipFree(d_result);

    return 0;
}